#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#define MAX_ITER 1000000
#define MAX 100 //maximum value of the matrix element
#define TOL 0.000001


void print_matrix(float ***N, int rows, int columns,int n) {

  int i, j;
  //print only 4*4 matrix.
  if (rows > 4)
    rows = 4;
  if (columns > 4)
    columns = 4;
  printf("Printing only first 4 results \n");
  for (i = 0; i < rows; i++) {
    printf("\n");
    for (j = 0; j < columns; j++) {
    
        printf("%lf \t", (*N)[i][j]); //row major accessing with red color.
    

    }
    
  }
  
}



// Generate a random float number with the maximum value of max
float rand_float(int max){
  return ((float)rand()/(float)(RAND_MAX)) * max;
}

// Allocate 2D matrix
void allocate_init_2Dmatrix(float ***mat,  int n, int m){
  int i, j;
  *mat = (float **) malloc(n * sizeof(float *));
  for(i = 0; i < n; i++) {
    (*mat)[i] = (float *)malloc(m * sizeof(float));
    for (j = 0; j < m; j++)
      (*mat)[i][j] = rand_float(MAX);
  }

}

// solver
__global__ void solver(float **matd, int n, int m, bool debug){
  float diff = 0, temp;
  int done = 0, cnt_iter = 0;
 int j= blockIdx.x*blockDim.x + threadIdx.x;
 int i= blockIdx.y*blockDim.y + threadIdx.y;
 
  while (!done && (cnt_iter < MAX_ITER)){
    diff = 0;
      if ((i < n - 1) && (j < m - 1) && (i > 0) && (j > 0)){
       temp = (matd)[i][j];
        (matd)[i][j] = 0.2 * ((matd)[i][j] + (matd)[i][j - 1] + (matd)[i - 1][j] + (matd)[i][j + 1] + (matd)[i + 1][j]);
        diff += abs((matd)[i][j] - temp);
  //      printf("diff:%f\n",diff);
      }

    if (diff/n/n < TOL)
      done = 1;
    cnt_iter ++;
  }
  if (debug){
    if (done)
      printf("Solver converged after %d iterations\n", cnt_iter);
    else
      printf("Solver not converged after %d iterations\n", cnt_iter);
  }
}

int main(int argc, char *argv[]) {
  int n;
  float **a,**ad;

  if (argc < 2) {
    printf("Call this program with two parameters: matrix_size communication \n");
    printf("\t matrix_size: Add 2 to a power of 2 (e.g. : 18, 1026)\n");

    exit(1);
  }

  n = atoi(argv[1]);
  int blocksize = atoi(argv[2]);
  float *temph[n];
  printf("Matrix size = %d\n", n);
  allocate_init_2Dmatrix(&a, n, n);

float gpu_elapsed_time_ms;

    // some events to count the execution time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // start to count execution time of GPU version
    hipEventRecord(start, 0);
// Allocate 2D array in Device
  hipMalloc((void **)&ad,n*sizeof(float *));
 for (int i = 0; i < n; i++){
        hipMalloc(&temph[i], n*sizeof(float));
}
  hipMemcpy(ad,temph,n*sizeof(float *),hipMemcpyHostToDevice);

 for (int i = 0; i < n; i++){
  hipMemcpy(temph[i],a[i],n*sizeof(float),hipMemcpyHostToDevice);
}


unsigned int grid_rows = (n + blocksize - 1) / blocksize;
  dim3 DimBlock(blocksize,blocksize);
  dim3 DimGrid(grid_rows,grid_rows);
solver<<<DimGrid, DimBlock>>>(ad, n, n,true);
hipMemcpy(a,ad,n*n*sizeof(float),hipMemcpyDeviceToHost);
hipFree(ad);

printf(">> Num of Block = %d | Block Dim = %d |Matrix size = %d\n", grid_rows, blocksize, n);
 hipDeviceSynchronize();
    // time counting terminate
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // compute time elapse on GPU computing
    hipEventElapsedTime(&gpu_elapsed_time_ms, start, stop);
     print_matrix(&a,n,n,n); 
    printf("Time elapsed on GPU: %f ms.\n\n",  gpu_elapsed_time_ms);
    
  return 0;
}