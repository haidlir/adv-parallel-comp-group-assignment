
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#define MAX_ITER 1000000
#define MAX 100 //maximum value of the matrix element
#define TOL 0.000001

// Generate a random float number with the maximum value of max
float rand_float(int max){
  return ((float)rand()/(float)(RAND_MAX)) * max;
}

// Allocate 2D matrix
void allocate_init_2Dmatrix(float ***mat,  int n, int m){
  int i, j;
  *mat = (float **) malloc(n * sizeof(float *));
  for(i = 0; i < n; i++) {
    (*mat)[i] = (float *)malloc(m * sizeof(float));
    for (j = 0; j < m; j++)
      (*mat)[i][j] = rand_float(MAX);
  }

}

// solver
__global__ void solver(float **matd, int n, int m, bool debug){
  float diff = 0, temp;
  int done = 0, cnt_iter = 0;
 int j= blockIdx.x*blockDim.x + threadIdx.x;
 int i= blockIdx.y*blockDim.y + threadIdx.y;
 if (debug) printf("valor de matd:%f\n",**matd);
  while (!done && (cnt_iter < MAX_ITER)){
    diff = 0;
      if ((i < n - 1) && (j < m - 1) && (i > 0) && (j > 0)){
       temp = (matd)[i][j];
        (matd)[i][j] = 0.2 * ((matd)[i][j] + (matd)[i][j - 1] + (matd)[i - 1][j] + (matd)[i][j + 1] + (matd)[i + 1][j]);
        diff += abs((matd)[i][j] - temp);
  //      printf("diff:%f\n",diff);
      }

    if (diff/n/n < TOL)
      done = 1;
    cnt_iter ++;
  }
  if (debug){
    if (done)
      printf("Solver converged after %d iterations\n", cnt_iter);
    else
      printf("Solver not converged after %d iterations\n", cnt_iter);
  }
}

int main(int argc, char *argv[]) {
  int n;
  float **a,**ad;
struct timeval start, end,start1, end1;
    double mtime, seconds, useconds,x,mtime1, seconds1, useconds1,y;
    gettimeofday(&start, NULL);
        dim3 DimGrid(10);
        dim3 DimBlock(128);
  if (argc < 2) {
    printf("Call this program with two parameters: matrix_size communication \n");
    printf("\t matrix_size: Add 2 to a power of 2 (e.g. : 18, 1026)\n");

    exit(1);
  }

  n = atoi(argv[1]);
  float *temph[n];
  printf("Matrix size = %d\n", n);
  allocate_init_2Dmatrix(&a, n, n);

// Allocate 2D array in Device
  hipMalloc((void **)&ad,n*sizeof(float *));
 for (int i = 0; i < n; i++){
        hipMalloc(&temph[i], n*sizeof(float));
}
  hipMemcpy(ad,temph,n*sizeof(float *),hipMemcpyHostToDevice);

 for (int i = 0; i < n; i++){
  hipMemcpy(temph[i],a[i],n*sizeof(float),hipMemcpyHostToDevice);
}

}